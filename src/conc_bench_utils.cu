#include "hip/hip_runtime.h"
/**
 * @file conc_bench_utils.cu
 * @date 8/03/2021
 * @author Mirco De Marchi
 * @brief Source of Concurrent Benchmark utils.
 */

/*
 * Apologies to whoever will have to read this code, I just discovered
 * precompiler macros and I went crazy with it..
 */
#include <iostream>
#include "Timer.cuh"
#include "CheckError.cuh"

#include "conc_bench_utils.cuh"

using namespace timer;

__global__ static void sum_gpu_left(float *matrix, const int N);
__global__ static void sum_gpu_right(float *matrix, const int N);

void matrix_supplier(std::vector<float> (&v))
{
    // -------------------------------------------------------------------------
    // MATRIX INITILIZATION
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);
    std::uniform_int_distribution<int> distribution(1, 100);

    for (int i = 0; i < N; i++)
    {
        float temp = distribution(generator);
        v.push_back(temp);
    }
}

void sum_cpu(std::vector<float> (&v), int sum_id) 
{
    float *arr = &v[0];
    if (sum_id % 2 != 0)
    {
        sum_cpu_right(arr, N);
    }
    else 
    {
        sum_cpu_left(arr, N);
    }
}

void sum_gpu(std::vector<float> (&v), float *d_matrix_device, int sum_id) 
{
    const int grid = N / BLOCK_SIZE_X;
    float *d_matrix = &v[0];
    // -------------------------------------------------------------------------
    // DEVICE INIT
    dim3 DimGrid(grid, 1, 1);
    if (N % grid)
        DimGrid.x++;
    dim3 DimBlock(BLOCK_SIZE_X, 1, 1);

    // -------------------------------------------------------------------------
    // EXECUTION
    if (sum_id % 2 != 0)
    {
#if COPY
        SAFE_CALL(hipMemcpy(d_matrix_device, d_matrix, N * sizeof(int),
                                hipMemcpyHostToDevice));
        sum_gpu_left<<<DimGrid, DimBlock>>>(d_matrix_device, N);
        CHECK_CUDA_ERROR
        SAFE_CALL(hipMemcpy(d_matrix, d_matrix_device, N * sizeof(int),
                                hipMemcpyDeviceToHost));
#else
        sum_gpu_left<<<DimGrid, DimBlock>>>(d_matrix, N);
#endif
#if UNIFIED
        // This macro includes hipDeviceSynchronize(), which makes the program
        // work on the data in lockstep
        CHECK_CUDA_ERROR
#endif
    }
    else
    {
#if COPY
        SAFE_CALL(hipMemcpy(d_matrix_device, d_matrix, N * sizeof(int),
                                hipMemcpyHostToDevice));
        sum_gpu_right<<<DimGrid, DimBlock>>>(d_matrix_device, N);
        CHECK_CUDA_ERROR
        SAFE_CALL(hipMemcpy(d_matrix, d_matrix_device, N * sizeof(int),
                                hipMemcpyDeviceToHost));
#else
        sum_gpu_right<<<DimGrid, DimBlock>>>(d_matrix, N);
#endif
#if UNIFIED
        CHECK_CUDA_ERROR
#endif
    }

#if ZEROCOPY
    // Synchronization needed to avoid race conditions (after the CPU and 
    // GPU have done their sides, we need to sync)
    CHECK_CUDA_ERROR
#endif
}

void init_gpu(float **d_matrix_device) 
{
#if ZEROCOPY
    hipSetDeviceFlags(hipDeviceMapHost);
#endif
    SAFE_CALL(hipMalloc(d_matrix_device, N * sizeof(float)));
}

void sum_cpu_right(float *d_matrix, const int N)
{
#if OPENMP
#pragma omp parallel for
#endif
    for (int j = N / 2; j < N; j++)
    {
        if (j % 2 == 0)
        {
            //__sync_fetch_and_add(&d_matrix[j], 1);
            for (int r = 0; r < 2; r++)
            {
                d_matrix[j] = sqrt(d_matrix[j] * (d_matrix[j] / 2.3));
            }
            // printf("cpu right: %d\n", j);
        }
    }
}

void sum_cpu_left(float *d_matrix, const int N)
{
#if OPENMP
#pragma omp parallel for
#endif
    for (int j = 0; j < N / 2; j++)
    {
        if (j % 2 != 0)
        {
            //__sync_fetch_and_add(&d_matrix[j], 1);
            for (int r = 0; r < 2; r++)
            {
                d_matrix[j] = sqrt(d_matrix[j] * (d_matrix[j] / 2.3));
            }
            // printf("cpu left: %d\n", j);
        }
    }
}

void sum_cpu_only(float *matrix)
{
#if CPU
    for (int i = 0; i < SUMS; i++)
    {
        if (i % 2 != 0)
        {
            for (int j = 0; j < N / 2; j++)
            {
                if (j % 2 != 0)
                {
                    float temp = 2.0 * sqrt(matrix[j] + matrix[j + N / 2]);
                    for (int f = 0; f < 2; f++)
                    {
                        temp /= float(f) + sqrt(3.14159265359 * temp) / 0.7;
                        temp *= 1.6;
                    }
                    matrix[j] = temp;
                }
            }
            for (int j = N / 2; j < N; j++)
            {
                if (j % 2 == 0)
                {
                    for (int r = 0; r < 2; r++)
                    {
                        matrix[j] = sqrt(matrix[j] * (matrix[j] / 2.3));
                    }
                }
            }
        }
        else
        {
            for (int j = N / 2; j < N; j++)
            {
                if (j % 2 == 0)
                {
                    float temp = 2.0 * sqrt(matrix[j] + matrix[j - N / 2]);
                    for (int f = 0; f < 2; f++)
                    {
                        temp /= float(f) + sqrt(3.14159265359 * temp) / 0.7;
                        temp *= 1.6;
                    }
                    matrix[j] = temp;
                }
            }
            for (int j = 0; j < N / 2; j++)
            {
                if (j % 2 != 0)
                {
                    for (int r = 0; r < 2; r++)
                    {
                        matrix[j] = sqrt(matrix[j] * (matrix[j] / 2.3));
                    }
                }
            }
        }
#if PRINT
        printf("RUN %d\n", i);
        printf("Values from index %d to %d\n", FROM_debug, TO_debug);
        printf("H: ");
        for (int i = FROM_debug; i < TO_debug; i++)
        {
            if (i % (N / 2) == 0)
                printf("| ");
            printf("%.2f ", matrix[i]);
        }
        printf("\n");
#endif
    }
#else
    for (int i = 0; i < SUMS; i++)
    {
        for (int j = 0; j < N / 2; j++)
        {
            if (j % 2 != 0)
            {
                float temp = 2.0 * sqrt(matrix[j] + matrix[j + N / 2]);
                for (int f = 0; f < 2; f++)
                {
                    temp /= float(f) + sqrt(3.14159265359 * temp) / 0.7;
                    temp *= 1.6;
                }
                matrix[j] = temp;
            }
        }
        for (int j = N / 2; j < N; j++)
        {
            if (j % 2 == 0)
            {
                float temp = 2.0 * sqrt(matrix[j] + matrix[j + N / 2]);
                for (int f = 0; f < 2; f++)
                {
                    temp /= float(f) + sqrt(3.14159265359 * temp) / 0.7;
                    temp *= 1.6;
                }
                matrix[j] = temp;
            }
        }
    }
#endif
}

__global__ static void sum_gpu_left(float *matrix, const int N)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N / 2)
    {
        if (row % 2 != 0)
        {
            float temp = 2.0 * sqrt(matrix[row] + matrix[row + N / 2]);
            for (int f = 0; f < 2; f++)
            {
                temp /= float(f) + sqrt(3.14159265359 * temp) / 0.7;
                temp *= 1.6;
            }
            matrix[row] = temp;
        }
    }
}

__global__ static void sum_gpu_right(float *matrix, const int N)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= N / 2 && row < N)
    {
        if (row % 2 == 0)
        {
            float temp = 2.0 * sqrt(matrix[row] + matrix[row - N / 2]);
            for (int f = 0; f < 2; f++)
            {
                temp /= float(f) + sqrt(3.14159265359 * temp) / 0.7;
                temp *= 1.6;
            }
            matrix[row] = temp;
        }
    }
}